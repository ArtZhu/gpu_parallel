/* file : search.cu
 * author : Tiane Zhu
 * date : Mar 30, 2017
 *
 * MAIN program 
 *
 * for
 * 1. a serial
 * 2. a parallel

 * implementation of the parallel search algorithm
 * 	ALGORITHM 4.1 in 
 * "An Introduction to Parallel Algorithms" - by Joseph Jaja
 *		p146 - ISBN 9-789201-548563
 */

#include "search.h"

// main
int main(int argc, char * argv[]) 
{
	setbuf(stdout, NULL);
	_init(argc, argv);

	if(verbose)
		printf("finding target : %d in array of length %d\n", target, X_len);

	hipError_t err_code[10];
	float gputime, cputime;

	hipSetDevice(0);
	hipDeviceReset();

	// X_len + 2 for the algorithm element at idx 0 and n + 1 (originally 1, 2, ..., n)
	err_code[0] = hipMalloc( &dev_X , X_size );
	err_code[1] = hipMalloc( &c , c_size );
	err_code[2] = hipMalloc( &q , q_size );
	for(int i=0; i<3; i++){ gerror(err_code[i]); }

	gerror(hipMemcpy(dev_X, host_X, X_size, hipMemcpyHostToDevice));

	unsigned int num_blocks = (1023 + num_threads) / 1024;
	unsigned int threads_per_block = num_threads > 1024 ? 1024 : num_threads;

	ret_idx = 10086;

	printf("launching %u blocks, %u threads per block.\n", num_blocks, threads_per_block);

	d->Dg = {num_blocks, 1, 1};
	d->Db = {threads_per_block, 1, 1};

	gstart();
	gpu_search(target, X_len, num_threads);
	gend(&gputime);
	printf("gputime : %f ms\n", gputime);
	gerror(hipGetLastError());
	gerror( hipDeviceSynchronize() );

	gerror(hipMemcpyFromSymbol(&ret_idx, HIP_SYMBOL(dev_ret), sizeof(int), 0, hipMemcpyDeviceToHost));
	printf("device idx = %d;\n", ret_idx);

	ret_idx = 10086;

	cstart();
	ret_idx = cpu_search(host_X + 1, X_len, target);
	cend(&cputime);
	printf("cputime : %f ms\n", cputime);
	printf("host idx = %d;\n", ret_idx);

	gerror(hipFree(dev_X));
	gerror(hipFree(c));
	gerror(hipFree(q));
	free(host_X);
}// main

char fname[80];
void _init(int argc, char ** argv)
{ 
	X_len = DEFAULT_ARRAY_LEN;
	num_threads = DEFAULT_NUM_THREADS;
	target = DEFAULT_TARGET;
	fname[0] = 0;

	int len_spec = 0;

	for(int i=1; i<argc; i++){
		switch(*argv[i]){
			case '-':
				switch(argv[i][1]){
					case 'v': 
						verbose = 1;
						break;
					case 'f':
						if(!len_spec){
							strcpy(fname, argv[++i]);
							len_spec = 1;
						}
						break;
					case 't':
						sscanf(argv[++i], "%d", &num_threads);
						break;
					case 'l':
						if(!len_spec){
							sscanf(argv[++i], "%d", &X_len);
							len_spec = 1;
						}
						break;
				}
				break;
			default:
				sscanf(argv[i], FMT, &target);
		}
	}

	X_size = (X_len + 2) * sizeof(number);
	c_size = (num_threads + 2) * sizeof(int);
	q_size = (num_threads + 2) * sizeof(int);

	_init_array(fname[0] != 0);

	prep_kernel();
}

void _init_array(int with_file)
{
	host_X = (number *) malloc(X_size);

	//not use file
	if(!with_file){
		for(number i=1; i<X_len+1; i++){
			host_X[i] = 2 * i;
		}
		return;
	}

	//use file
	FILE * fp;
	printf("array file : \"%s\"", fname);

	if(!(fp = fopen(fname, "r"))){
		printf(" does not exist.\n");
		exit(1);
	}

	if(fscanf(fp, "%d", &X_len) < 1){
		printf(" stats broken.\n");
		exit(1);
	}

	printf("\n");

	for(int i=0; i<X_len; i++){
		if(fscanf(fp, FMT, host_X + i) != 1){
			printf(" missing the %dth number.\n", i);
			exit(1);
		}
		if(verbose)
			printf(FMT, host_X[i]);
	}
	if(verbose) printf("\n");

}
