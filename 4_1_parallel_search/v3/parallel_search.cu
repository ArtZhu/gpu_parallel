#include "hip/hip_runtime.h"
/* file : parallel_search.cu
 * author : Tiane Zhu
 * date : Mar 23, 2017
 *
 * this program is an implementation of the parallel search algorithm
 * 	ALGORITHM 4.1 in 
 * "An Introduction to Parallel Algorithms" - by Joseph Jaja
 *		p146 - ISBN 9-789201-548563
 */

#include "parallel_search.h"

///////////////////////////////////////////////////////////
// Input to the algorithm //
// X 										-- strictly ordered array
// y (target) 					-- target
// p (num_threads) 			-- num_processor
// j (tid) 							-- processor idx
///////////////////////////////////////////////////////////
// Output 
// i (ret) 							-- X[i] <= y < x[i+1]
//		[ i is initialized to -1 , since it has only non-neg values
//			i non-neg => i set ]
///////////////////////////////////////////////////////////
/* kernel strictly following algorithm */
// additional inputs
volatile int * c;
// c										-- c array from 0 to p+1
volatile int * q;
// q										-- q array from 0 to p+1
volatile __device__ int l;
// l										
volatile __device__ int r;
// r
// n is the number of elements
// this function needs iter_flag to be initialized to -1
__global__ void search(number * X, int n, number target, volatile int * c, volatile int * q, int num_threads, int * dev_ret){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	// ATOMIC
	// for comparison and conditionally setting global iter flag.
	int local_iter = 0;
	int * mutex = half_iter_signals + tid;

	if(tid == 0)
		printf("%p, %p\n", half_iter_signals, mutex);

	tid += 1; // so that idx starts from 1

	if(tid > n) return; // safety

	//1.
	// come back and add the first atomic flag here.
	if(tid == 1){
		l = 0;
		r = n + 1;
		X[0] = INT_MIN;
		X[n + 1] = INT_MAX;
		c[0] = 0;
		c[num_threads + 1] = 1;


		*dev_ret = -1; // for thread termination purpose
		//init iter_flag
		atomicExch(&iter_flag, 0);

	}

	//init signals
	atomicExch(mutex, 0);

#ifdef PRETTY_PRINT
	if(tid == 1){
		//for(int i=0; i<n+2; i++) printf("%d ", X[i]); 
		//printf("\n");
		//printf("| q0 | q1 | q2 | q3 | c0 | c1 | c2 | c3 | l  | r  |\n");
		printf("%d %d\n", r, l);
	}
#endif

	//sync
	//__syncthreads();

	//2.
	while(atomicCAS(&iter_flag, 0, 0) != 0); // second arg doesn't matter here

	if(tid == 2)
		printf("4\n");
	
	//while(r - l > num_threads){

		if(tid == 1){
			q[0] = l;
			q[num_threads + 1] = r;
		}

	if(tid == 2)
		printf("5\n");

		q[tid] = l + tid * ((r - l) / (num_threads + 1));

		//sync -- use r, l, p;
		//		 -- set q
		//__syncthreads();
		//__threadfence();

		if(target == X[q[tid]]){
			*dev_ret = q[tid] - 1; // so that ret idx starts from 0
			// can i return here???
			// no
			//return;
		}
		else{
			if(target > X[q[tid]])
				c[tid] = 0;
			else 
				c[tid] = 1;
		}

	if(tid == 2)
		printf("6\n");

		//sync -- use X, q, target
		//     -- set l, r, c
		//__syncthreads();
		//__threadfence();

		// if ret has been set, return, a replacement for the "return" in the conditional statement;
		// put this in the end, and use atomic flag = iteration 
		//		atomic flag signal end of while iteration.
		//     it also signals l, r has been set and then check this 
		// problematic
		//if(*dev_ret >= 0){
//#ifdef PRETTY_PRINT
//			if(tid == 1)
//				printf("dev ret0 : %d\n", *dev_ret);
//#endif
//			return;
		//}

		
		//mark
		__threadfence();
		//guarantees, tid-1 read the value already.
	
	if(tid == 2)
		printf("7\n");

		if(tid != 1)
			while(atomicCAS(mutex, 0, 1) != 0);

		__threadfence();

	if(tid == 2)
		printf("8\n");

		//guarantees, tid+1 set the value already
		if(tid != n)
			while(atomicCAS(mutex + 1, 1, 0) != 1);
	/*


		// whoever sets l,r  should let other threads know that 
		//			next iteration is ready.
		// above this, thread no race condition
		// problematic part 
		if(c[tid] < c[tid + 1]){
			l = q[tid];
			r = q[tid + 1];

			__threadfence();
			
			atomicCAS(&iter_flag, local_iter, local_iter+1);
		}

		if(tid == 1 && c[0] < c[1]){
			l = q[0];
			r = q[1];

			__threadfence();
			
			atomicCAS(&iter_flag, local_iter, local_iter+1);
		}

		//sync -- use q, c, tid
		//		 -- set l, r
		//__syncthreads();

		++local_iter;
		//*iter_flag ok here?
		while(atomicCAS(&iter_flag, 0, local_iter) != local_iter); // second arg doesn't matter here

		if(*dev_ret >= 0){
#ifdef PRETTY_PRINT
			if(tid == 1)
				printf("dev ret0 : %d\n", *dev_ret);
#endif
			return;
		}

#ifdef PRETTY_PRINT
		if(tid == 1)
			printf("%d %d\n", r, l);
			//printf("|%4d|%4d|%4d|%4d|%4d|%4d|%4d|%4d|%4d|%4d|\n", q[0], q[1], q[2], q[3], c[0], c[1], c[2], c[3], l, r);
#endif

	}

	
	if(tid > r - l) return;

	if(target == X[l+tid]){
		//*dev_ret = l + tid - 1; // so that ret idx starts from 0
		atomicCAS(dev_ret, -1, l + tid - 1);
	}
	else if(target > X[l+tid]){
		c[tid] = 0;
	}
	else{
		c[tid] = 1;
	}

	//mark
	__threadfence();
	//set flag.
	atomicExch(mutex, 1);

	//guarantees, tid-1 set the flag already
	if(tid != 1)
		while(atomicCAS(mutex - 1, 1, 0) != 1);

#ifdef PRETTY_PRINT
	printf("dev ret1 : %d\n", *dev_ret);
#endif
	if(*dev_ret >= 0)
		return;


	// problematic part
	if(c[tid-1] < c[tid])
		atomicCAS(dev_ret, -1, l + tid - 1 - 1); // so that ret idx starts from 0
#ifdef PRETTY_PRINT
	printf("dev ret2 : %d\n", *dev_ret);
#endif
	*/
}

// main
int main(int argc, char * argv[]) 
{
	setbuf(stdout, NULL);

	_init(argc, argv);

	if(verbose)
		printf("finding target : %d in array of length %d\n", target, X_len);

	hipError_t err_code[10];
	float gputime, cputime;
	int ret_idx, * dev_ret;
	
	hipSetDevice(0);
	hipDeviceReset();

	// X_len + 2 for the algorithm element at idx 0 and n + 1 (originally 1, 2, ..., n)
	gerror(hipMalloc( &dev_X , X_size ));
	gerror(hipMalloc( &c , c_size ));
	gerror(hipMalloc( &q , q_size ));
	gerror(hipMalloc( &dev_ret , sizeof(int) ));
	gerror(hipMalloc( &host_half_iter_signals_ptr, num_threads * sizeof(int)));
	//gerror(err_code[4] = hipMalloc( &half_iter_signals, num_threads * sizeof(int)));
	//err_code[4] = hipMalloc( &half_iter_signals, num_threads * sizeof(int));

	printf("1\n");

	gerror(hipMemcpyToSymbol(HIP_SYMBOL(half_iter_signals), &host_half_iter_signals_ptr, sizeof(int *), 0, hipMemcpyHostToDevice));

	printf("2\n");
	//use it as a tmp var
	ret_idx = -1;

	gerror(hipMemcpyToSymbol(HIP_SYMBOL(iter_flag), &ret_idx, sizeof(int), 0, hipMemcpyHostToDevice));

	printf("3\n");
	
	gerror(hipMemcpy(dev_X, host_X, X_size, hipMemcpyHostToDevice));

	unsigned int num_blocks = (1023 + num_threads) / 1024;
	unsigned int threads_per_block = num_threads > 1024 ? 1024 : num_threads;

	ret_idx = 10086;

	printf("launching %u blocks, %u threads per block.\n", num_blocks, threads_per_block);

	d->Dg = {num_blocks, 1, 1};
	d->Db = {threads_per_block, 1, 1};
	gstart();
	search<<<d->Dg, d->Db>>>(dev_X, X_len, target, c, q, num_threads, dev_ret);
	gend(&gputime);
	printf("gputime : %f ms\n", gputime);
	gerror( hipGetLastError());
	gerror( hipDeviceSynchronize() );

	gerror(hipMemcpy(&ret_idx, dev_ret, sizeof(int), hipMemcpyDeviceToHost));
	printf("device idx = %d;\n", ret_idx);

	ret_idx = 10086;

	cstart();
	ret_idx = cpu_search(host_X + 1, X_len, target);
	cend(&cputime);
	printf("cputime : %f ms\n", cputime);
	printf("host idx = %d;\n", ret_idx);

	gerror(hipFree(dev_X));
	gerror(hipFree((void *) c));
	gerror(hipFree((void *) q));
	gerror(hipFree(host_half_iter_signals_ptr));
	gerror(hipFree(dev_ret));
	free(host_X);
}

char fname[80];
void _init(int argc, char ** argv)
{ 
	X_len = DEFAULT_ARRAY_LEN;
	num_threads = DEFAULT_NUM_THREADS;
	target = DEFAULT_TARGET;
	fname[0] = 0;

	int len_spec = 0;

	for(int i=1; i<argc; i++){
		switch(*argv[i]){
			case '-':
				switch(argv[i][1]){
					case 'v': 
						verbose = 1;
						break;
					case 'f':
						if(!len_spec){
							strcpy(fname, argv[++i]);
							len_spec = 1;
						}
						break;
					case 't':
						sscanf(argv[++i], "%d", &num_threads);
						break;
					case 'l':
						if(!len_spec){
							sscanf(argv[++i], "%d", &X_len);
							len_spec = 1;
						}
						break;
				}
				break;
			default:
				sscanf(argv[i], FMT, &target);
		}
	}

	X_size = (X_len + 2) * sizeof(number);
	c_size = (num_threads + 2) * sizeof(int);
	q_size = (num_threads + 2) * sizeof(int);

	_init_array(fname[0] != 0);
	
	prep_kernel();
}

void _init_array(int with_file)
{
	host_X = (number *) malloc(X_size);

	//not use file
	if(!with_file){
		for(number i=1; i<X_len+1; i++){
			host_X[i] = 2 * i;
		}
		return;
	}
	
	//use file
	FILE * fp;
	printf("array file : \"%s\"", fname);

	if(!(fp = fopen(fname, "r"))){
		printf(" does not exist.\n");
		exit(1);
	}

	if(fscanf(fp, "%d", &X_len) < 1){
		printf(" stats broken.\n");
		exit(1);
	}

	printf("\n");

	for(int i=0; i<X_len; i++){
		if(fscanf(fp, FMT, host_X + i) != 1){
			printf(" missing the %dth number.\n", i);
			exit(1);
		}
		if(verbose)
			printf(FMT, host_X[i]);
	}
	if(verbose) printf("\n");

}
