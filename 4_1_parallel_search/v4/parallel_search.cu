#include "hip/hip_runtime.h"
/* file : parallel_search.cu
 * author : Tiane Zhu
 * date : Mar 23, 2017
 *
 * this program is an implementation of the parallel search algorithm
 * 	ALGORITHM 4.1 in 
 * "An Introduction to Parallel Algorithms" - by Joseph Jaja
 *		p146 - ISBN 9-789201-548563
 */

#include "parallel_search.h"

///////////////////////////////////////////////////////////
// Input to the algorithm //
// X 										-- strictly ordered array
// y (target) 					-- target
// p (num_threads) 			-- num_processor
// j (tid) 							-- processor idx
///////////////////////////////////////////////////////////
// Output 
// i (ret) 							-- X[i] <= y < x[i+1]
//		[ i is initialized to -1 , since it has only non-neg values
//			i non-neg => i set ]
///////////////////////////////////////////////////////////
/* kernel strictly following algorithm */
// additional inputs
volatile int * c;
// c										-- c array from 0 to p+1
volatile int * q;
// q										-- q array from 0 to p+1
volatile __device__ int l;
// l										
volatile __device__ int r;
// r
// n is the number of elements
// this function needs iter_flag to be initialized to -1
__global__ void search(number * X, int n, number target, volatile int * c, volatile int * q, int num_threads, int * dev_ret){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	// ATOMIC
	// for comparison and conditionally setting global iter flag.
	int local_iter = 0;
	int * mutex = half_iter_signals + tid;

	tid += 1; // so that idx starts from 1

	if(tid > n) return; // safety, this needs to be fixed, tid+1 part

	//init signals
	//atomicExch(mutex, 0);
	*mutex = 0;

	//1.
	// come back and add the first atomic flag here.
	if(tid == 1){
		l = 0;
		r = n + 1;
		X[0] = INT_MIN;
		X[n + 1] = INT_MAX;
		c[0] = 0;
		c[num_threads + 1] = 1;


		*dev_ret = -1; // for thread termination purpose
		//init iter_flag
#ifdef PRETTY_PRINT
		printf("INITIAL: r: %d; l: %d\n", r, l);
#endif

#ifdef PRETTY_PRINT
		printf("num threads: %d\n", num_threads);
#endif

		__threadfence();

		atomicExch(&iter_flag, 0);
	}


	//2.
	while(atomicCAS(&iter_flag, 0, 0) != 0); // second arg doesn't matter here

#ifdef PRETTY_PRINT
	if(*mutex != 0) printf("%d mutex not 0\n", tid);
#endif


	while(r - l > num_threads){

		if(tid == 1){
			q[0] = l;
			q[num_threads + 1] = r;
		}


		int qtid = l + tid * ((r - l) / (num_threads + 1));
		q[tid] = qtid;

		if(target == X[qtid]){
			*dev_ret = qtid - 1; // so that ret idx starts from 0
		}
		else{
			if(target > X[qtid])
				c[tid] = 0;
			else 
				c[tid] = 1;
		}

		//mark
		__threadfence();
		//guarantees, tid-1 read the value already.

		if(tid != 1)
			while(atomicCAS(mutex, 0, 1) != 0);

		//guarantees, tid+1 set the value already
		if(tid != num_threads)
			while(atomicCAS(mutex + 1, 1, 0) != 1);


		int iter_inc = 0;
		// whoever sets l,r  should let other threads know that 
		//			next iteration is ready.
		if(c[tid] < c[tid + 1]){
			l = q[tid];
			r = q[tid + 1];

			__threadfence();

			//atomicCAS(&iter_flag, local_iter, local_iter+1);
			iter_inc = 1;
		}

		if(tid == 1 && c[0] < c[1]){
			l = q[0];
			r = q[1];

			__threadfence();

			//atomicCAS(&iter_flag, local_iter, local_iter+1);
			iter_inc = 1;
		}

		atomicCAS(&iter_flag, local_iter, local_iter+iter_inc);

		++local_iter;

		//*iter_flag ok here?
		while(atomicCAS(&iter_flag, -1, local_iter) != local_iter); // second arg doesn't matter here

		if(*dev_ret >= 0)
			return;
		
	}



	if(tid <= r - l){
		if(target == X[l+tid]){
			//*dev_ret = l + tid - 1; // so that ret idx starts from 0
			atomicCAS(dev_ret, -1, l + tid - 1);
		}
		else if(target > X[l+tid]){
			c[tid] = 0;
		}
		else{
			c[tid] = 1;
		}
	}

	//mark
	__threadfence();
	//set flag.
	atomicExch(mutex, 1);

	//guarantees, tid-1 set the flag already
	if(tid != 1)
		while(atomicCAS(mutex - 1, 1, 0) != 1);

#ifdef PRETTY_PRINT
	if(tid == 1)
		printf("dev ret1 : %d\n", *dev_ret);
#endif

	if(*dev_ret >= 0)
		return;

	if(tid <= r - l){

		// problematic part
		if(c[tid-1] < c[tid])
			*dev_ret = l + tid - 1 - 1; // so that ret idx starts from 0

#ifdef PRETTY_PRINT
		printf("dev ret2 : %d\n", *dev_ret);
#endif

	}
}

// main
int main(int argc, char * argv[]) 
{
	setbuf(stdout, NULL);

	_init(argc, argv);

	if(verbose)
		printf("finding target : %d in array of length %d\n", target, X_len);

	float gputime, cputime;
	int ret_idx, * dev_ret;
	
	hipSetDevice(0);
	hipDeviceReset();

	// X_len + 2 for the algorithm element at idx 0 and n + 1 (originally 1, 2, ..., n)
	gerror(hipMalloc( &dev_X , X_size ));
	gerror(hipMalloc( &c , c_size ));
	gerror(hipMalloc( &q , q_size ));
	gerror(hipMalloc( &dev_ret , sizeof(int) ));
	gerror(hipMalloc( &host_half_iter_signals_ptr, num_threads * sizeof(int)));

	gerror(hipMemcpyToSymbol(HIP_SYMBOL(half_iter_signals), &host_half_iter_signals_ptr, sizeof(int *), 0, hipMemcpyHostToDevice));

	//use it as a tmp var
	ret_idx = -1;

	gerror(hipMemcpyToSymbol(HIP_SYMBOL(iter_flag), &ret_idx, sizeof(int), 0, hipMemcpyHostToDevice));

	gerror(hipMemcpy(dev_X, host_X, X_size, hipMemcpyHostToDevice));

	unsigned int num_blocks = (1023 + num_threads) / 1024;
	unsigned int threads_per_block = num_threads > 1024 ? 1024 : num_threads;

	ret_idx = 10086;

	printf("launching %u blocks, %u threads per block.\n", num_blocks, threads_per_block);

	d->Dg = {num_blocks, 1, 1};
	d->Db = {threads_per_block, 1, 1};
	gstart();
	search<<<d->Dg, d->Db>>>(dev_X, X_len, target, c, q, num_threads, dev_ret);
	gend(&gputime);
	printf("gputime : %f ms\n", gputime);
	gerror( hipGetLastError());
	gerror( hipDeviceSynchronize() );

	gerror(hipMemcpy(&ret_idx, dev_ret, sizeof(int), hipMemcpyDeviceToHost));
	printf("device idx = %d;\n", ret_idx);

	ret_idx = 10086;

	cstart();
	ret_idx = cpu_search(host_X + 1, X_len, target);
	cend(&cputime);
	printf("cputime : %f ms\n", cputime);
	printf("host idx = %d;\n", ret_idx);

	gerror(hipFree(dev_X));
	gerror(hipFree((void *) c));
	gerror(hipFree((void *) q));
	gerror(hipFree(host_half_iter_signals_ptr));
	gerror(hipFree(dev_ret));
	free(host_X);
}

char fname[80];
void _init(int argc, char ** argv)
{ 
	X_len = DEFAULT_ARRAY_LEN;
	num_threads = DEFAULT_NUM_THREADS;
	target = DEFAULT_TARGET;
	fname[0] = 0;

	int len_spec = 0;

	for(int i=1; i<argc; i++){
		switch(*argv[i]){
			case '-':
				switch(argv[i][1]){
					case 'v': 
						verbose = 1;
						break;
					case 'f':
						if(!len_spec){
							strcpy(fname, argv[++i]);
							len_spec = 1;
						}
						break;
					case 't':
						sscanf(argv[++i], "%d", &num_threads);
						break;
					case 'l':
						if(!len_spec){
							sscanf(argv[++i], "%d", &X_len);
							len_spec = 1;
						}
						break;
				}
				break;
			default:
				sscanf(argv[i], FMT, &target);
		}
	}

	X_size = (X_len + 2) * sizeof(number);
	c_size = (num_threads + 2) * sizeof(int);
	q_size = (num_threads + 2) * sizeof(int);

	_init_array(fname[0] != 0);
	
	prep_kernel();
}

void _init_array(int with_file)
{
	host_X = (number *) malloc(X_size);

	//not use file
	if(!with_file){
		for(number i=1; i<X_len+1; i++){
			host_X[i] = 2 * i;
		}
		return;
	}
	
	//use file
	FILE * fp;
	printf("array file : \"%s\"", fname);

	if(!(fp = fopen(fname, "r"))){
		printf(" does not exist.\n");
		exit(1);
	}

	if(fscanf(fp, "%d", &X_len) < 1){
		printf(" stats broken.\n");
		exit(1);
	}

	printf("\n");

	for(int i=0; i<X_len; i++){
		if(fscanf(fp, FMT, host_X + i) != 1){
			printf(" missing the %dth number.\n", i);
			exit(1);
		}
		if(verbose)
			printf(FMT, host_X[i]);
	}
	if(verbose) printf("\n");

}
