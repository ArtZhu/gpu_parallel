#include "hip/hip_runtime.h"
/* file : parallel_search.cu
 * author : Tiane Zhu
 * date : Mar 23, 2017
 *
 * this program is an implementation of the parallel search algorithm
 * 	ALGORITHM 4.1 in 
 * "An Introduction to Parallel Algorithms" - by Joseph Jaja
 *		p146 - ISBN 9-789201-548563
 */

#include "parallel_search.h"

///////////////////////////////////////////////////////////
// Input to the algorithm //
// X 										-- strictly ordered array
// y (target) 					-- target
// p (num_threads) 			-- num_processor
// j (tid) 							-- processor idx
///////////////////////////////////////////////////////////
// Output 
// i (ret) 							-- X[i] <= y < x[i+1]
//		[ i is initialized to -1 , since it has only non-neg values
//			i non-neg => i set ]
///////////////////////////////////////////////////////////
/* kernel strictly following algorithm */
// additional inputs
int * c;
// c										-- c array from 0 to p+1
int * q;
// q										-- q array from 0 to p+1
int * l;
// l	must be allocated to num_blocks size
int * r;
// r  must be allocated to num_blocks size
volatile int * dev_ret;
// dev_ret  must be allocated to num_blocks size
// n is the number of elements
__device__ void search(number * X, int n, number target, int * c, int * q, int num_threads, volatile int * dev_ret, int * l, int * r){
	int tid = threadIdx.x;

	X += n * blockIdx.x;
	l += blockIdx.x;
	r += blockIdx.x;
	dev_ret += blockIdx.x;
	c += blockIdx.x * (blockDim.x + 2);
	q += blockIdx.x * (blockDim.x + 2);

	tid += 1; // so that idx starts from 1

	if(tid > n) return; // safety

	//1.
  // initialize this part outside kernel
	if(tid == 1){
		*l = 0;
		*r = n + 1;
		c[0] = 0;
		c[num_threads + 1] = 1;

		*dev_ret = -2; // for thread termination purpose
	}

#ifdef PRETTY_PRINT
	if(tid == 1)
		printf("%d : %d %d\n", blockIdx.x, *l, *r);
#endif

	//sync
	__syncthreads();

	//2.

	int count = 0;
	while(*r - *l > num_threads){

		if(tid == 1){
			q[0] = *l;
			q[num_threads + 1] = *r;
		}

		q[tid] = *l + tid * ((*r - *l) / (num_threads + 1));

		//sync -- use r, l, p;
		//		 -- set q
		__syncthreads();

		if(target == X[q[tid]]){
			*dev_ret = q[tid] - 1; // so that ret idx starts from 0
			// can i return here???
			// no
			//return;
		}
		else{
			if(target > X[q[tid]])
				c[tid] = 0;
			else 
				c[tid] = 1;
		}

		//sync -- use X, q, target
		//     -- set l, r, c
		__syncthreads();
		// if ret has been set, return, a replacement for the "return" in the conditional statement;
		if(*dev_ret >= -1){
#ifdef PRETTY_PRINT
		if(tid == 1)
			printf("%d : dev_ret0 %d\n", blockIdx.x, *dev_ret);
#endif
			return;
		}


		if(c[tid] < c[tid + 1]){
			*l = q[tid];
			*r = q[tid + 1];
		}


		if(tid == 1 && c[0] < c[1]){
			*l = q[0];
			*r = q[1];
		}

		//sync -- use q, c, tid
		//		 -- set l, r
		__syncthreads();


#ifdef PRETTY_PRINT
		if(tid == 1)
			printf("iter %d, block %d : %d %d\n", count++, blockIdx.x, *l, *r);
#endif

	}

#ifdef PRETTY_PRINT
		if(tid == 1)
			printf("%d : dev_ret1 %d\n", blockIdx.x, *dev_ret);
#endif
	if(tid > *r - *l) return;

	if(target == X[*l+tid]){
		*dev_ret = *l + tid - 1; // so that ret idx starts from 0
	}
	else if(target > X[*l+tid]){
		c[tid] = 0;
	}
	else{
		c[tid] = 1;
	}

#ifdef PRETTY_PRINT
		if(tid == 1)
			printf("%d : dev_ret2 %d\n", blockIdx.x, *dev_ret);
#endif
	if(*dev_ret >= -1)
		return;

	if(c[tid-1] < c[tid])
		*dev_ret = *l + tid - 1 - 1; // so that ret idx starts from 0

#ifdef PRETTY_PRINT
		if(tid == 1)
			printf("%d : dev_ret3 %d\n", blockIdx.x, *dev_ret);
#endif

}

__device__ void fix(volatile int * dev_ret, int dev_ret_len, int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	//while(dev_ret[tid] == -2) ;

	if(tid < dev_ret_len){
		int idx = dev_ret[tid];
		if(idx != -2 && idx != -1){
			dev_ret[0] = idx + n * tid;
		}
	}
}

__global__ void search_main(number * X, int n, number target, int * c, int * q, int num_threads, volatile int * dev_ret, int * l, int * r, int dev_ret_len)
{
	// doesn't work for non-pow 2
	int tmp_n = n / dev_ret_len;

	/*
	if(threadIdx.x == 0){
		printf("array length : %d\n", n);
		printf("tmp_n : %d\n dev_ret_len = %d\n", tmp_n, dev_ret_len);
	}
	*/

	num_threads = num_threads > 1024 ? 1024 : num_threads;

	search(X, tmp_n, target, c, q, num_threads, dev_ret, l, r);

	/*
	if(blockIdx.x == 0 && threadIdx.x == 0){
		printf("[ ");
		for(int i=0; i<dev_ret_len; i++){
			printf("%d ", dev_ret[i]);
		}
		printf("]\n");
	}
	*/

	
	fix(dev_ret, dev_ret_len, tmp_n);

	__threadfence();

	if(threadIdx.x + blockIdx.x * blockDim.x == 0)
	printf("gpu found : %d\n", *dev_ret);

}

// main
int main(int argc, char * argv[]) 
{
	setbuf(stdout, NULL);
	_init(argc, argv);

	if(verbose)
		printf("finding target : %d in array of length %d\n", target, X_len);

	hipError_t err_code[10];
	float gputime, cputime;
	int ret_idx, * dev_ret;
	
	hipSetDevice(0);
	hipDeviceReset();

	unsigned int num_blocks = (1023 + num_threads) / 1024;
	unsigned int threads_per_block = num_threads > 1024 ? 1024 : num_threads;

	c_size = (2 * num_blocks + num_threads) * sizeof(int);
	q_size = (2 * num_blocks + num_threads) * sizeof(int);

	// X_len + 2 for the algorithm element at idx 0 and n + 1 (originally 1, 2, ..., n)
	err_code[0] = hipMalloc( &dev_X , X_size );
	err_code[1] = hipMalloc( &c , c_size );
	err_code[2] = hipMalloc( &q , q_size );
	err_code[3] = hipMalloc( &dev_ret , sizeof(volatile int) * num_blocks);
	err_code[4] = hipMalloc( &l , sizeof(int) * num_blocks );
	err_code[5] = hipMalloc( &r , sizeof(int) * num_blocks );
	for(int i=0; i<6; i++){ gerror(err_code[i]); }

	gerror(hipMemcpy(dev_X, host_X, X_size, hipMemcpyHostToDevice));

	ret_idx = 10086;

	printf("launching %u blocks, %u threads per block.\n", num_blocks, threads_per_block);

	d->Dg = {num_blocks, 1, 1};
	d->Db = {threads_per_block, 1, 1};
	gstart();
	search_main<<<d->Dg, d->Db>>>(dev_X, X_len, target, c, q, num_threads, dev_ret, l, r, num_blocks);
	gend(&gputime);
	printf("gputime : %f ms\n", gputime);
	gerror(hipGetLastError());
	gerror( hipDeviceSynchronize() );

	gerror(hipMemcpy(&ret_idx, dev_ret, sizeof(int), hipMemcpyDeviceToHost));
	printf("device idx = %d;\n", ret_idx);

	ret_idx = 10086;

	cstart();
	ret_idx = cpu_search(host_X + 1, X_len, target);
	cend(&cputime);
	printf("cputime : %f ms\n", cputime);
	printf("host idx = %d;\n", ret_idx);

	gerror(hipFree(dev_X));
	gerror(hipFree(c));
	gerror(hipFree(q));
	gerror(hipFree(dev_ret));
	gerror(hipFree(l));
	gerror(hipFree(r));
	free(host_X);
}

char fname[80];
void _init(int argc, char ** argv)
{ 
	X_len = DEFAULT_ARRAY_LEN;
	num_threads = DEFAULT_NUM_THREADS;
	target = DEFAULT_TARGET;
	fname[0] = 0;

	int len_spec = 0;

	for(int i=1; i<argc; i++){
		switch(*argv[i]){
			case '-':
				switch(argv[i][1]){
					case 'v': 
						verbose = 1;
						break;
					case 'f':
						if(!len_spec){
							strcpy(fname, argv[++i]);
							len_spec = 1;
						}
						break;
					case 't':
						sscanf(argv[++i], "%d", &num_threads);
						break;
					case 'l':
						if(!len_spec){
							sscanf(argv[++i], "%d", &X_len);
							len_spec = 1;
						}
						break;
				}
				break;
			default:
				sscanf(argv[i], FMT, &target);
		}
	}

	X_size = (X_len + 2) * sizeof(number);

	_init_array(fname[0] != 0);
	
	prep_kernel();
}

void _init_array(int with_file)
{
	host_X = (number *) malloc(X_size);

	host_X[0] = INT_MIN;
	host_X[X_len+1] = INT_MAX;
	//not use file
	if(!with_file){
		for(number i=1; i<X_len+1; i++){
			host_X[i] = 2 * i;
		}
		return;
	}
	
	//use file
	FILE * fp;
	printf("array file : \"%s\"", fname);

	if(!(fp = fopen(fname, "r"))){
		printf(" does not exist.\n");
		exit(1);
	}

	if(fscanf(fp, "%d", &X_len) < 1){
		printf(" stats broken.\n");
		exit(1);
	}

	printf("\n");

	for(int i=0; i<X_len; i++){
		if(fscanf(fp, FMT, host_X + i) != 1){
			printf(" missing the %dth number.\n", i);
			exit(1);
		}
		if(verbose)
			printf(FMT, host_X[i]);
	}
	if(verbose) printf("\n");

}
