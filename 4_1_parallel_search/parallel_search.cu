#include "hip/hip_runtime.h"
/* file : parallel_search.cu
 * author : Tiane Zhu
 * date : Mar 23, 2017
 *
 * this program is an implementation of the parallel search algorithm
 * 	ALGORITHM 4.1 in 
 * "An Introduction to Parallel Algorithms" - by Joseph Jaja
 *		p146 - ISBN 9-789201-548563
 */

#include "parallel_search.h"

///////////////////////////////////////////////////////////
// Input to the algorithm //
// X 										-- strictly ordered array
// y (target) 					-- target
// p (num_threads) 			-- num_processor
// j (tid) 							-- processor idx
///////////////////////////////////////////////////////////
// Output 
// i (ret) 							-- X[i] <= y < x[i+1]
//		[ i is initialized to -1 , since it has only non-neg values
//			i non-neg => i set ]
///////////////////////////////////////////////////////////
/* kernel strictly following algorithm */
// additional inputs
int * c;
// c										-- c array from 0 to p+1
int * q;
// q										-- q array from 0 to p+1
__device__ int l;
// l										
__device__ int r;
// r
// n is the number of elements
__global__ void search(number * X, int n, number target, int * c, int * q, int num_threads, int * dev_ret){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	tid += 1; // so that idx starts from 1

	//1.
	if(tid == 1){
		l = 0;
		r = n + 1;
		X[0] = INT_MIN;
		X[n + 1] = INT_MAX;
		c[0] = 0;
		c[num_threads + 1] = 1;

		*dev_ret = -1; // for thread termination purpose
	}

#ifdef PRETTY_PRINT
	if(tid == 1){
		for(int i=0; i<n+2; i++) printf("%d ", X[i]); 
		printf("\n");
		printf("| q0 | q1 | q2 | q3 | c0 | c1 | c2 | c3 | l  | r  |\n");
	}
#endif

	//sync
	__syncthreads();

	//2.
	int count = 0;

	while(r - l > num_threads){

		if(tid == 1){
			q[0] = l;
			q[num_threads + 1] = r;
		}

		q[tid] = l + tid * ((r - l) / (num_threads + 1));

		//sync -- use r, l, p;
		//		 -- set q
		__syncthreads();

		if(target == X[q[tid]]){
			*dev_ret = q[tid];
			// can i return here???
			// no
			//return;
		}
		else{
			if(target > X[q[tid]])
				c[tid] = 0;
			else 
				c[tid] = 1;
		}

		//sync -- use X, q, target
		//     -- set l, r, c
		__syncthreads();
		// if ret has been set, return, a replacement for the "return" in the conditional statement;
		if(*dev_ret > 0){
#ifdef PRETTY_PRINT
			if(tid == 1)
				printf("dev ret0 : %d\n", *dev_ret);
#endif
			return;
		}

		if(c[tid] < c[tid + 1]){
			l = q[tid];
			r = q[tid + 1];
		}

		if(tid == 1 && c[0] < c[1]){
			l = q[0];
			r = q[1];
		}

		//sync -- use q, c, tid
		//		 -- set l, r
		__syncthreads();

#ifdef PRETTY_PRINT
		if(tid == 1)
			printf("|%4d|%4d|%4d|%4d|%4d|%4d|%4d|%4d|%4d|%4d|\n", q[0], q[1], q[2], q[3], c[0], c[1], c[2], c[3], l, r);
#endif

		if(++count > 10){
			printf("oops\n");
			return;
		}
	}

	if(tid > r - l) return;

	if(target == X[l+tid]){
		*dev_ret = l + tid;
	}
	else if(target > X[l+tid]){
		c[tid] = 0;
	}
	else{
		c[tid] = 1;
	}

#ifdef PRETTY_PRINT
	printf("dev ret1 : %d\n", *dev_ret);
#endif
	if(*dev_ret > 0)
		return;

	if(c[tid-1] < c[tid])
		*dev_ret = l + tid - 1;
#ifdef PRETTY_PRINT
	printf("dev ret2 : %d\n", *dev_ret);
#endif
}

// main
int main(int argc, char * argv[]) 
{
	setbuf(stdout, NULL);
	_init(argc, argv);

	hipError_t err_code[10];
	float gputime, cputime;
	int ret_idx, * dev_ret;
	
	hipSetDevice(0);
	hipDeviceReset();

	// X_len + 2 for the algorithm element at idx 0 and n + 1 (originally 1, 2, ..., n)
	err_code[0] = hipMalloc( &dev_X , X_size );
	err_code[1] = hipMalloc( &c , c_size );
	err_code[2] = hipMalloc( &q , q_size );
	err_code[3] = hipMalloc( &dev_ret , sizeof(int) );
	for(int i=0; i<4; i++){ gerror(err_code[i]); }

	gerror(hipMemcpy(dev_X, host_X, X_size, hipMemcpyHostToDevice));

	unsigned int num_blocks = num_threads > 1024 ? num_threads / 1024 + 1 : 1;
	unsigned int threads_per_block = num_threads > 1024 ? 1024 : num_threads;

	ret_idx = 10086;

	printf("launching %u blocks, %u threads per block.\n", num_blocks, threads_per_block);

	d->Dg = {num_blocks, 1, 1};
	d->Db = {threads_per_block, 1, 1};
	gstart();
	//search<<<d->Dg, d->Db>>>(dev_X, X_len, target, c, q, num_threads, dev_ret);
	gend(&gputime);
	printf("gputime : %f ms\n", gputime);
	gerror(hipGetLastError());
	gerror( hipDeviceSynchronize() );

	gerror(hipMemcpy(&ret_idx, dev_ret, sizeof(int), hipMemcpyDeviceToHost));
	printf("device idx = %d;\n", ret_idx);

	cstart();
	ret_idx = cpu_search(host_X + 1, X_len, target);
	cend(&cputime);
	printf("cputime : %f ms\n", cputime);
	printf("host idx = %d;\n", ret_idx);

	gerror(hipFree(dev_X));
	gerror(hipFree(c));
	gerror(hipFree(q));
	free(host_X);
}

char fname[80];
void _init(int argc, char ** argv)
{ 
	X_len = DEFAULT_ARRAY_LEN;
	num_threads = DEFAULT_NUM_THREADS;
	target = DEFAULT_TARGET;
	fname[0] = 0;

	int len_spec = 0;

	for(int i=1; i<argc; i++){
		switch(*argv[i]){
			case '-':
				switch(argv[i][1]){
					case 'v': 
						verbose = 1;
						break;
					case 'f':
						if(!len_spec){
							strcpy(fname, argv[++i]);
							len_spec = 1;
						}
						break;
					case 't':
						sscanf(argv[++i], "%d", &num_threads);
						break;
					case 'l':
						if(!len_spec){
							sscanf(argv[++i], "%d", &X_len);
							len_spec = 1;
						}
						break;
				}
				break;
			default:
				sscanf(argv[i], FMT, &target);
		}
	}

	X_size = (X_len + 2) * sizeof(number);
	c_size = (num_threads + 2) * sizeof(int);
	q_size = (num_threads + 2) * sizeof(int);

	_init_array(fname[0] != 0);
}

void _init_array(int with_file)
{
	host_X = (number *) malloc(X_size);

	//not use file
	if(!with_file){
		for(number i=1; i<X_len+1; i++){
			host_X[i] = 2 * i;
		}
		return;
	}
	
	//use file
	FILE * fp;
	printf("array file : \"%s\"", fname);

	if(!(fp = fopen(fname, "r"))){
		printf(" does not exist.\n");
		exit(1);
	}

	if(fscanf(fp, "%d", &X_len) < 1){
		printf(" stats broken.\n");
		exit(1);
	}

	printf("\n");

	for(int i=0; i<X_len; i++){
		if(fscanf(fp, FMT, host_X + i) != 1){
			printf(" missing the %dth number.\n", i);
			exit(1);
		}
		if(verbose)
			printf(FMT, host_X[i]);
	}
	if(verbose) printf("\n");

}
