#include "hip/hip_runtime.h"
/* file : parallel_search.cu
 * author : Tiane Zhu
 * date : Mar 23, 2017
 *
 * this program is an implementation of the parallel search algorithm
 * 	ALGORITHM 4.1 in 
 * "An Introduction to Parallel Algorithms" - by Joseph Jaja
 *		p146 - ISBN 9-789201-548563
 */

#include "parallel_search.h"

///////////////////////////////////////////////////////////
// Input to the algorithm //
// X 										-- strictly ordered array
// y (target) 					-- target
// p (num_threads) 			-- num_processor
// j (tid) 							-- processor idx
///////////////////////////////////////////////////////////
// Output 
// i (ret) 							-- X[i] <= y < x[i+1]
//		[ i is initialized to -1 , since it has only non-neg values
//			i non-neg => i set ]
///////////////////////////////////////////////////////////
/* kernel strictly following algorithm */
// additional inputs
int * c;
// c										-- c array from 0 to p+1
int * q;
// q										-- q array from 0 to p+1
__device__ int l;
// l										
__device__ int r;
// r
__device__ int iter = 0;
// n is the number of elements
__global__ void search(number * X, int n, number target, int * c, int * q, int num_threads, int * dev_ret){

	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	tid += 1; // so that idx starts from 1

#ifdef PRETTY_PRINT
	if(tid == 1)
		printf("KERNEL : \narray length : %d, target : %d, num_threads : %d\n", n, target, num_threads);
#endif

	//if(tid > n) return; // safety

	//1.
	if(tid == 1 && iter == 0){
		l = 0;
		r = n + 1;
		X[0] = INT_MIN;
		X[n + 1] = INT_MAX;
		c[0] = 0;
		c[num_threads + 1] = 1;

		*dev_ret = -1; // for thread termination purpose
	}

	if(tid == 1)
		iter++;

#ifdef PRETTY_PRINT
	if(tid == 1){
		if(n <= 32){
			for(int i=0; i<n+2; i++) printf("%d ", X[i]); 
		}
		//printf("\n");
		//printf("| q0 | q1 | q2 | q3 | c0 | c1 | c2 | c3 | l  | r  |\n");
		printf("1. r : %d ; l : %d\n", r, l);
	}
#endif

	//sync -- 
	//		 -- set r, l, X, c, dev_ret, tid
	__syncthreads();

	//2.

	// if statement below as replacement
	//while(r - l > num_threads){
	int set_ret = (r - l <= num_threads);

	if(tid == 1){
		q[0] = l;
		q[num_threads + 1] = r;
	}

	q[tid] = l + tid * ((r - l) / (num_threads + 1));

	//sync -- use r, l, p, tid, num_threads;
	//		 -- set q
	__syncthreads();

	if(target == X[q[tid]]){
		*dev_ret = q[tid] - 1; // so that ret idx starts from 0
		// can i return here???
		// no
		//return;
	}
	else{
		if(target > X[q[tid]])
			c[tid] = 0;
		else 
			c[tid] = 1;
	}

	//sync -- use X, q, target, tid
	//     -- set dev_ret, c
	__syncthreads();

	// if ret has been set, return, a replacement for the "return" in the conditional statement;
	if(*dev_ret >= 0){
#ifdef PRETTY_PRINT
		if(tid == 1)
			printf("dev ret0 : %d\n", *dev_ret);
#endif
		return;
	}


	if(c[tid] < c[tid + 1]){
		l = q[tid];
		r = q[tid + 1];
		printf("tid : %d setting r, l to be %d %d\n", tid, r, l);
	}

	//sync -- use dev_ret, q, c, tid
	//		 -- set l, r
	__syncthreads();


	if(tid == 1 && c[0] < c[1]){
		l = q[0];
		r = q[1];
	}

	//sync -- use dev_ret, q, c, tid
	//		 -- set l, r
	__syncthreads();


#ifdef PRETTY_PRINT
	if(tid == 1){
		printf("r : %d ; l : %d\n", r, l);
		printf("c[%d] = %d, c[%d] = %d, c[%d] = %d\n", 1023, c[1023], 1024, c[1024], 1025, c[1025]);
		//printf("|%4d|%4d|%4d|%4d|%4d|%4d|%4d|%4d|%4d|%4d|\n", q[0], q[1], q[2], q[3], c[0], c[1], c[2], c[3], l, r);
	}
#endif

	//} //while(r - l > num_threads){
		if(set_ret){

			if(tid > r - l){ 
				//corresponds with the next syncthreads();
				__syncthreads();
				return;
			}

			if(target == X[l+tid]){
				*dev_ret = l + tid - 1; // so that ret idx starts from 0
			}
			else if(target > X[l+tid]){
				c[tid] = 0;
			}
			else{
				c[tid] = 1;
			}

			// sync -- use l, X, tid, target
			//			-- set dev_ret, c
			__syncthreads();

#ifdef PRETTY_PRINT
			printf("dev ret1 : %d\n", *dev_ret);
#endif
			if(*dev_ret >= 0)
				return;

			if(c[tid-1] < c[tid])
				*dev_ret = l + tid - 1 - 1; // so that ret idx starts from 0
#ifdef PRETTY_PRINT
			printf("dev ret2 : %d\n", *dev_ret);
#endif
		}
}

// main
int main(int argc, char * argv[]) 
{
	setbuf(stdout, NULL);
	_init(argc, argv);

	if(verbose)
		printf("finding target : %d in array of length %d\n", target, X_len);

	hipError_t err_code[10];
	float gputime, cputime;
	int ret_idx, * dev_ret;
	
	hipSetDevice(0);
	hipDeviceReset();

	// X_len + 2 for the algorithm element at idx 0 and n + 1 (originally 1, 2, ..., n)
	err_code[0] = hipMalloc( &dev_X , X_size );
	err_code[1] = hipMalloc( &c , c_size );
	err_code[2] = hipMalloc( &q , q_size );
	err_code[3] = hipMalloc( &dev_ret , sizeof(int) );
	for(int i=0; i<4; i++){ gerror(err_code[i]); }

	gerror(hipMemcpy(dev_X, host_X, X_size, hipMemcpyHostToDevice));

	unsigned int num_blocks = (1023 + num_threads) / 1024;
	unsigned int threads_per_block = num_threads > 1024 ? 1024 : num_threads;

	ret_idx = 10086;

	printf("launching %u blocks, %u threads per block.\n", num_blocks, threads_per_block);

	d->Dg = {num_blocks, 1, 1};
	d->Db = {threads_per_block, 1, 1};
	gstart();
	do{
		search<<<d->Dg, d->Db>>>(dev_X, X_len, target, c, q, num_threads, dev_ret);
		gerror(hipMemcpy(&ret_idx, dev_ret, sizeof(int), hipMemcpyDeviceToHost));
	}while(ret_idx < 0);
	gend(&gputime);
	printf("gputime : %f ms\n", gputime);
	gerror(hipGetLastError());
	gerror( hipDeviceSynchronize() );

	//gerror(hipMemcpy(&ret_idx, dev_ret, sizeof(int), hipMemcpyDeviceToHost));
	printf("device idx = %d;\n", ret_idx);

	ret_idx = 10086;

	cstart();
	ret_idx = cpu_search(host_X + 1, X_len, target);
	cend(&cputime);
	printf("cputime : %f ms\n", cputime);
	printf("host idx = %d;\n", ret_idx);

	gerror(hipFree(dev_X));
	gerror(hipFree(c));
	gerror(hipFree(q));
	gerror(hipFree(dev_ret));
	free(host_X);
}// main

char fname[80];
void _init(int argc, char ** argv)
{ 
	X_len = DEFAULT_ARRAY_LEN;
	num_threads = DEFAULT_NUM_THREADS;
	target = DEFAULT_TARGET;
	fname[0] = 0;

	int len_spec = 0;

	for(int i=1; i<argc; i++){
		switch(*argv[i]){
			case '-':
				switch(argv[i][1]){
					case 'v': 
						verbose = 1;
						break;
					case 'f':
						if(!len_spec){
							strcpy(fname, argv[++i]);
							len_spec = 1;
						}
						break;
					case 't':
						sscanf(argv[++i], "%d", &num_threads);
						break;
					case 'l':
						if(!len_spec){
							sscanf(argv[++i], "%d", &X_len);
							len_spec = 1;
						}
						break;
				}
				break;
			default:
				sscanf(argv[i], FMT, &target);
		}
	}

	X_size = (X_len + 2) * sizeof(number);
	c_size = (num_threads + 2) * sizeof(int);
	q_size = (num_threads + 2) * sizeof(int);

	_init_array(fname[0] != 0);
	
	prep_kernel();
}

void _init_array(int with_file)
{
	host_X = (number *) malloc(X_size);

	//not use file
	if(!with_file){
		for(number i=1; i<X_len+1; i++){
			host_X[i] = 2 * i;
		}
		return;
	}
	
	//use file
	FILE * fp;
	printf("array file : \"%s\"", fname);

	if(!(fp = fopen(fname, "r"))){
		printf(" does not exist.\n");
		exit(1);
	}

	if(fscanf(fp, "%d", &X_len) < 1){
		printf(" stats broken.\n");
		exit(1);
	}

	printf("\n");

	for(int i=0; i<X_len; i++){
		if(fscanf(fp, FMT, host_X + i) != 1){
			printf(" missing the %dth number.\n", i);
			exit(1);
		}
		if(verbose)
			printf(FMT, host_X[i]);
	}
	if(verbose) printf("\n");

}
