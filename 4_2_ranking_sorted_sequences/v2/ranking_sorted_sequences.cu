#include "hip/hip_runtime.h"
/* filename : ranking_sorted_sequences.cu
 * author: Tiane Zhu
 * date : Mar 26, 2017
 *
 * this file contains an O(1) ranking parallel algorithm
 *
 * this program is an implementation of the ranking sorted sequences
 *		ALGORITHM 4.2 in 
 * "An Introduction to Parallel Algorithms" - by Joseph Jaja
 *			p150 - ISBN 9-789201-548563
 */

////
// Input : 	A = (a1, ... , an)
// 					B = (b1, ... , bm)
//			sqm = sqrt(m)
////
// Output : rank(B : A)
////
// begin
// 		1. If m < 4, then rank the elements of B 
//			 							by applying Alg 4_1 with p = n
//								 then exit
//		2. Concurrently rank b_sqm, b_2sqm, ..., bm in A 
//										by applying Alg 4_1 with p = sqrt(n)
//			 Let 	j[i] = rank(b_isqm : A)
//						j[0] = 0
//		3. For 0 <= i <= sqm - 1,
//					let B_i = ( b_isqm+1, ... , b_(i+1)sqm - 1 )
//					let A_i = ( a_j[i]+1, ... , a_j[i+1] )
//			 If j[i] == j[i+1], then
//					set rank(B_i : A_i) = (0, ... , 0)
//			 else
//					recurse compute rank(B_i : A_i)
//		4. Let 1 <= k <= m be an arbitrary index not multiple of sqm
//			 Let i = floor( k / sqm )
//			 rank(b_k : A) = j[i] + rank(b_k : A_i)
// end
////


#include "ranking_sorted_sequences.h"

int main(){
	int n = 8;
	number A[] = {-5, 0, 3, 4, 17, 18, 24, 28};
	int m = 4;
	number B[] = {1, 2, 15, 21};
	number ret[m];

	number * dev_A, * dev_B, *dev_ret;
	
	hipMalloc(&dev_A, n * sizeof(number));
	hipMalloc(&dev_B, m * sizeof(number));
	hipMalloc(&dev_ret, m * sizeof(number));

	hipMemcpy(dev_A, A, n * sizeof(number), hipMemcpyHostToDevice);
	hipMemcpy(dev_B, B, m * sizeof(number), hipMemcpyHostToDevice);

	int num_threads = 1024;

	ranking<<<1, 1024>>>(dev_A, n, dev_B, dev_ret, m, num_threads);

	hipDeviceSynchronize();

	hipMemcpy(ret, dev_ret, m * sizeof(number), hipMemcpyDeviceToHost);
	printf("\n GPU RANKING : [ ");
	for(int i=0; i<m; i++)
		printf("%d ", ret[i]);
	printf("]\n");

	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_ret);

	cpu_ranking(A, n, B, ret, m);

	printf("\n CPU RANKING : [ ");
	for(int i=0; i<m; i++)
		printf("%d ", ret[i]);
	printf("]\n");

}

//A  = INT_MIN, a0, a1, ..., an, INT_MAX

__global__ void ranking(number * A, int n, number * B, int * ret, int m, int num_threads)
{
	int i, dev_ret;
	number target;

// 		1. If m < 4, then rank the elements of B 
//			 							by applying Alg 4_1 with p = n
//								 then exit
	if(m < 4){
		for(i = 0; i<m; i++){

			target = B[i];
			search(A, n, target, num_threads, &dev_ret);
			ret[i] = *((int *) &search_rank);
			if(threadIdx.x == 0){
				printf("%d\n", dev_ret);
			}

			__syncthreads();
		}
		return;
	}

//		2. Concurrently rank b_sqm, b_2sqm, ..., bm in A 
//										by applying Alg 4_1 with p = sqrt(n)
//			 Let 	j[i] = rank(b_isqm : A)
//						j[0] = 0
//		3. For 0 <= i <= sqm - 1,
//					let B_i = ( b_isqm+1, ... , b_(i+1)sqm - 1 )
//					let A_i = ( a_j[i]+1, ... , a_j[i+1] )
//			 If j[i] == j[i+1], then
//					set rank(B_i : A_i) = (0, ... , 0)
//			 else
//					recurse compute rank(B_i : A_i)
//		4. Let 1 <= k <= m be an arbitrary index not multiple of sqm
//			 Let i = floor( k / sqm )
//			 rank(b_k : A) = j[i] + rank(b_k : A_i)
// end
}
